#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;
using std::endl;

typedef long long ll;


__global__ void dkernel(long int* g_mat, long int *g_ans, long int* g_filter, int m, int n, int k){

    extern __shared__ long int fil_cpy[];


    int i = blockIdx.x;
    int j = threadIdx.x;

    if (threadIdx.x == 0){
        for(int l = 0; l < k * k; l++){
            fil_cpy[l] = g_filter[l];
        }
    }

    __syncthreads();
    
    for(int p = i - k/2; p < i - k/2 + k; p++){
        for(int q = j - k/2; q < j - k/2 + k; q++){
            if (p >= 0 && p <= m - 1 && q >= 0 && q <= n - 1){
                g_ans[i * n + j] += (g_mat[p * n + q] * fil_cpy[(p - i + k/2) * k + q - j + k/2]);
            }
        }
    }
}


int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];


    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/
    long int* g_mat = new long int[m * n];
    long int* g_filter = new long int[k * k];
    long int* g_ans = new long int[m * n];

    hipMalloc(&g_mat,m * n * sizeof(long int));
    hipMalloc(&g_filter,k * k * sizeof(long int));
    hipMalloc(&g_ans,m * n * sizeof(long int));

    hipMemcpy(g_mat, h_mat, m * n * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(g_filter, h_filter, k * k * sizeof(long int), hipMemcpyHostToDevice);

    
    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch

    dkernel<<<m, n, k * k * sizeof(long int)>>>(g_mat, g_ans, g_filter, m, n, k);

    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch

    hipDeviceSynchronize();

    hipMemcpy(h_ans, g_ans, m * n * sizeof(long int), hipMemcpyDeviceToHost);
    
    
    
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */
 

    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}
